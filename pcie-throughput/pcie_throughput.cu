
#include <hip/hip_runtime.h>
#include <stdio.h>
// #include <cuda.h>
#include <nvml.h>

#define dugCheck(result_t, success, format, getString, operation) \
    do {                                                          \
        result_t status = (operation);                            \
        if (status != success) {                                  \
            printf("NVML ERROR: %s (%d): %s\n", __FILE__, __LINE__, getString(status));                            \
        }                                                         \
    } while (0)

#define dugNvmlCheck(operation) dugCheck(nvmlReturn_t, NVML_SUCCESS, "%s", nvmlErrorString, operation)


int main() { 

    nvmlInit();

    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    char name[32];
    nvmlDeviceGetName(device, name, 32);
    printf("Device name: %s\n\n", name);


    unsigned int tx, rx; // throughput in KB/s
    dugNvmlCheck(nvmlDeviceGetPcieThroughput(device, NVML_PCIE_UTIL_TX_BYTES, &tx));
    dugNvmlCheck(nvmlDeviceGetPcieThroughput(device, NVML_PCIE_UTIL_RX_BYTES, &rx));

    printf("PCIe TX throughput: %d KB/s\n", tx);
    printf("PCIe RX throughput: %d KB/s\n", rx);

    printf("\n");

    printf("PCIe TX throughput: %f GB/s\n", float(tx) / (1024.0 * 1024.0));
    printf("PCIe RX throughput: %f GB/s\n", float(rx) / (1024.0 * 1024.0));


    return 0;
}