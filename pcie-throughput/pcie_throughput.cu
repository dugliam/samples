#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <nvml.h>

int main() { 

    nvmlInit();

    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    char name[32];
    nvmlDeviceGetName(device, name, 32);
    printf("Device name: %s\n\n", name);


    unsigned int tx, rx; // throughput in KB/s
    nvmlDeviceGetPcieThroughput(0, NVML_PCIE_UTIL_TX_BYTES, &tx);
    nvmlDeviceGetPcieThroughput(0, NVML_PCIE_UTIL_RX_BYTES, &rx);

    printf("PCIe TX throughput: %d KB/s\n", tx);
    printf("PCIe RX throughput: %d KB/s\n", rx);

    printf("\n");

    printf("PCIe TX throughput: %f GB/s\n", float(tx) / (1024.0 * 1024.0));
    printf("PCIe RX throughput: %f GB/s\n", float(rx) / (1024.0 * 1024.0));


    return 0;
}