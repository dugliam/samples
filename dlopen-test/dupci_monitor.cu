#include <dlfcn.h>
#include <nvml.h>
#include <stdlib.h>
#include <stdio.h>

#include "dupci_monitor.h"

#define dugCheck(result_t, success, format, getString, operation) \
    do {                                                          \
        result_t status = (operation);                            \
        if (status != success) {                                  \
            printf("NVML ERROR: %s (%d): %s\n", __FILE__, __LINE__, getString(status));                            \
        }                                                         \
    } while (0)

#define dugNvmlCheck(operation) dugCheck(nvmlReturn_t, NVML_SUCCESS, "%s", nvmlErrorString, operation)
// #define dugNvmlCheck(operation) operation

struct dugPCIMonitorContext {
    void *handle;
    nvmlReturn_t (*duNvmlInit_v2)(void);
    const char* (*duNvmlErrorString)(nvmlReturn_t);
    nvmlReturn_t (*duNvmlDeviceGetHandleByIndex_v2)(unsigned int, nvmlDevice_t*);
    nvmlReturn_t (*duNvmlDeviceGetPcieThroughput)(nvmlDevice_t, nvmlPcieUtilCounter_t, unsigned int*);
    nvmlReturn_t (*duNvmlDeviceGetName)(nvmlDevice_t, char*, unsigned int);
};

// returns bytes/s tx and rx over previous 20ms
static void pci_throughput_sample(struct dugPCIMonitorContext *ctx, int device, unsigned long* tx_bytes, unsigned long* rx_bytes) { 
    nvmlDevice_t dev_handle, dev_handle2;

    int *h_a;
    int *d_a;
    int N = 1000000000;

    h_a = (int*)malloc(N*sizeof(int));
    hipMalloc(&d_a, N*sizeof(int));

    hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

    // nvmlDeviceGetHandleByIndex_v2(device, &dev_handle);
    (*(ctx->duNvmlDeviceGetHandleByIndex_v2))(device, &dev_handle);

    char name[32];
    nvmlDeviceGetName(dev_handle, name, 32);
    printf("    (LD) Device name: %s\n", name);
    (*(ctx->duNvmlDeviceGetName))(dev_handle, name, 32);
    printf("(DLOPEN) Device name: %s\n\n", name);

    unsigned int tx, rx; // throughput in KB/s
    dugNvmlCheck(nvmlDeviceGetPcieThroughput(dev_handle, NVML_PCIE_UTIL_TX_BYTES, &tx));
    dugNvmlCheck(nvmlDeviceGetPcieThroughput(dev_handle, NVML_PCIE_UTIL_RX_BYTES, &rx));

    unsigned int tx_kb, rx_kb;
    dugNvmlCheck((*(ctx->duNvmlDeviceGetPcieThroughput))(dev_handle, NVML_PCIE_UTIL_TX_BYTES, &tx_kb));
    dugNvmlCheck((*(ctx->duNvmlDeviceGetPcieThroughput))(dev_handle, NVML_PCIE_UTIL_RX_BYTES, &rx_kb));

    printf("    (LD) PCIe TX throughput: %d KB/s\n", tx);
    printf("    (LD) PCIe RX throughput: %d KB/s\n", rx);

    printf("(DLOPEN) PCIe TX throughput: %d KB/s\n", tx_kb);
    printf("(DLOPEN) PCIe RX throughput: %d KB/s\n", rx_kb);

    *tx_bytes = tx_kb * 1024;
    *rx_bytes = rx_kb * 1024;
}

// estimates of total tx/rx pcie throughput by extrapolating with the actual elapsed time
void pci_throughput_estimate(struct dugPCIMonitorContext *ctx, int device, unsigned long *tx_bytes, unsigned long *rx_bytes, unsigned long time_us) {
	unsigned long tx_bytes_sample, rx_bytes_sample;
	
	pci_throughput_sample(ctx, device, &tx_bytes_sample, &rx_bytes_sample);

	*tx_bytes = (unsigned long)(tx_bytes_sample * ((float)time_us / 1000000.0));
	*rx_bytes = (unsigned long)(rx_bytes_sample * ((float)time_us / 1000000.0));
}

static void* getDLSymbol(void *handle, char *symbol) { 
    void *sym;
    char *error;
    int success = 0;
    sym = dlsym(handle, symbol);
    if ((error = dlerror()) != NULL)  {
        printf("%s\n", error);
    }
    return sym;
}

struct dugPCIMonitorContext* dugPCIMonitorContextInit() { 
    struct dugPCIMonitorContext *ctx = (struct dugPCIMonitorContext*)malloc(sizeof(struct dugPCIMonitorContext));

    if(!ctx)
        return NULL;

    ctx->handle = dlopen("libnvidia-ml.so", RTLD_LAZY);

    if (!ctx->handle) {
        printf("%s\n", dlerror());
        return NULL;
    } else { 
        if((*(void**) (&(ctx->duNvmlDeviceGetHandleByIndex_v2)) = getDLSymbol(ctx->handle, "nvmlDeviceGetHandleByIndex_v2")) == NULL) { 
            dlclose(ctx->handle);
            return NULL;
        }
        if((*(void**) (&(ctx->duNvmlDeviceGetPcieThroughput)) = getDLSymbol(ctx->handle, "nvmlDeviceGetPcieThroughput")) == NULL) {             
            dlclose(ctx->handle);
            return NULL;
        }
        if((*(void**) (&(ctx->duNvmlInit_v2)) = getDLSymbol(ctx->handle, "nvmlInit_v2")) == NULL) {             
            dlclose(ctx->handle);
            return NULL;
        }
        if((*(void**) (&(ctx->duNvmlDeviceGetName)) = getDLSymbol(ctx->handle, "nvmlDeviceGetName")) == NULL) {             
            dlclose(ctx->handle);
            return NULL;
        }
    }
    // initialise NVML
    (*(ctx->duNvmlInit_v2))();
    return ctx;
}

void dugPCIMonitorContextFini(struct dugPCIMonitorContext* ctx) { 
    if(!ctx)
        return;
    
    if(ctx->handle)
        dlclose(ctx->handle);
    
    free(ctx);
}

int main() {
    printf("Acquiring ctx\n");
    unsigned long tx_bytes, rx_bytes;

    struct dugPCIMonitorContext* ctx = dugPCIMonitorContextInit();

    if(!ctx) { 
        printf("Couldn't load PCIMonitorContext\n");
    } else {
        printf("Loaded PCIMonitorContext\n");
        pci_throughput_sample(ctx, 0, &tx_bytes, &rx_bytes);
        printf("  (MAIN) TX throughput: %ld B/s\n", tx_bytes);
        printf("  (MAIN) RX throughput: %ld B/s\n", rx_bytes);
    }
    dugPCIMonitorContextFini(ctx);

    return 0;
}