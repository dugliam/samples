
#include <hip/hip_runtime.h>
#include <cstdio>
#define STR_HELPER(x) #x
#define STR(x) STR_HELPER(x)

__device__ void print_arch(){
    const char my_compile_time_arch[] = STR(__CUDA_ARCH__);
    printf("__CUDA_ARCH__: %s\n", my_compile_time_arch);
}
__global__ void example()
{
   print_arch();
}

int main(){
    example<<<1,1>>>();
    hipDeviceSynchronize();
}