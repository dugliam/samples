#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <cmath>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "block_io.hpp"
#include "common.hpp"
#include "random.hpp"

template<class FFT>
__launch_bounds__(FFT::max_threads_per_block) __global__
    void block_fft_kernel(typename FFT::value_type* data, unsigned int repeats) {
    using complex_type = typename FFT::value_type;
    extern __shared__ complex_type shared_mem[];

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFT>::load(data, thread_data, local_fft_id);

    // Execute FFT
    #pragma unroll 1
    for (unsigned int i = 0; i < repeats; i++) {
        FFT().execute(thread_data, shared_mem);
    }

    // Save results
    example::io<FFT>::store(thread_data, data, local_fft_id);
}

template<bool UseGraphs = false>
struct measure {
    // Returns execution time in ms
    template<unsigned int WarmUpRuns, typename Kernel>
    static float execution(Kernel&& kernel, hipStream_t stream) {
        hipEvent_t startEvent, stopEvent;
        CUDA_CHECK_AND_EXIT(hipEventCreate(&startEvent));
        CUDA_CHECK_AND_EXIT(hipEventCreate(&stopEvent));
        CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

        for (size_t i = 0; i < WarmUpRuns; i++) {
            kernel();
        }
        CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

        CUDA_CHECK_AND_EXIT(hipEventRecord(startEvent, stream));
        kernel();
        CUDA_CHECK_AND_EXIT(hipEventRecord(stopEvent, stream));
        CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

        float time;
        CUDA_CHECK_AND_EXIT(hipEventElapsedTime(&time, startEvent, stopEvent));
        CUDA_CHECK_AND_EXIT(hipEventDestroy(startEvent));
        CUDA_CHECK_AND_EXIT(hipEventDestroy(stopEvent));
        return time;
    }
};

template<class FFTBase, unsigned int S /* Size */, unsigned int EPT, unsigned int FPB = 1, bool UseSuggested = false>
void benchmark_block_fft(const hipStream_t& stream, bool verbose = false) {
    using namespace cufftdx;

    // Create complete FFT description, only now we can query EPT and suggested FFTs per block
    using FFT_complete = decltype(FFTBase() + Size<S>());

    static constexpr unsigned int inside_repeats = 4000;
    static constexpr unsigned int kernel_repeats = 1;
    static constexpr unsigned int warm_up_runs   = 1;

    static constexpr unsigned int fft_size            = S;
    static constexpr unsigned int elements_per_thread = UseSuggested ? FFT_complete::elements_per_thread : EPT;
    static constexpr unsigned int ffts_per_block      = UseSuggested ? FFT_complete::suggested_ffts_per_block : FPB;

    using FFT = decltype(FFT_complete() + ElementsPerThread<elements_per_thread>() + FFTsPerBlock<ffts_per_block>());
    using complex_type = typename FFT::value_type;

    int blocks_per_multiprocessor = 0;
    // [TODO] returns 0 for 4k kernel
    CUDA_CHECK_AND_EXIT(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocks_per_multiprocessor,
                                                             block_fft_kernel<FFT>,
                                                             FFT::block_dim.x * FFT::block_dim.y * FFT::block_dim.z,
                                                             FFT::shared_memory_size));

    hipDeviceProp_t device_properties;
    CUDA_CHECK_AND_EXIT(hipGetDeviceProperties(&device_properties, 0));
    unsigned int cuda_blocks = blocks_per_multiprocessor * device_properties.multiProcessorCount;
    unsigned int input_size  = ffts_per_block * fft_size * cuda_blocks;

    // Host data
    std::vector<complex_type> input =
        example::get_random_complex_data<typename complex_type::value_type>(input_size, -10, 10);

    // Device data
    complex_type* device_buffer;
    auto          size_bytes = input.size() * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&device_buffer, size_bytes));
    // Copy host to device
    CUDA_CHECK_AND_EXIT(hipMemcpy(device_buffer, input.data(), size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Measure performance of N trials
    double ms_n = measure<>::execution<warm_up_runs>(
        [&]() {
            for (unsigned int i = 0; i < kernel_repeats; i++) {
                block_fft_kernel<FFT>
                    <<<cuda_blocks, FFT::block_dim, FFT::shared_memory_size, stream>>>(device_buffer, inside_repeats);
            }
        },
        stream);

    // Copy host to device
    CUDA_CHECK_AND_EXIT(hipMemcpy(device_buffer, input.data(), size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Measure performance of 2*N trials
    double ms_n2 = measure<>::execution<warm_up_runs>(
        [&]() {
            for (unsigned int i = 0; i < kernel_repeats; i++) {
                block_fft_kernel<FFT><<<cuda_blocks, FFT::block_dim, FFT::shared_memory_size, stream>>>(
                    device_buffer, 2 * inside_repeats);
            }
        },
        stream);

    CUDA_CHECK_AND_EXIT(hipFree(device_buffer));

    // Time for N repeats without overhead
    auto   time_n = ms_n2 - ms_n;
    double gflops = 1.0 * kernel_repeats * inside_repeats * ffts_per_block * cuda_blocks * 5.0 * fft_size *
                    (std::log(fft_size) / std::log(2)) / time_n / 1000000.0;
    if (verbose) {
        std::cout << "FFT size: " << fft_size << std::endl;
        std::cout << "FFTs elements per thread: " << FFT::elements_per_thread << std::endl;
        std::cout << "FFTs per block: " << ffts_per_block << std::endl;
        std::cout << "CUDA blocks: " << cuda_blocks << std::endl;
        std::cout << "Blocks per multiprocessor: " << blocks_per_multiprocessor << std::endl;
        std::cout << "FFTs run: " << ffts_per_block * cuda_blocks << std::endl;
        std::cout << "Shared memory: " << FFT::shared_memory_size << std::endl;
        std::cout << "Avg Time [ms_n]: " << time_n / (inside_repeats * kernel_repeats) << std::endl;
        std::cout << "Time (all) [ms_n]: " << time_n << std::endl;
        std::cout << "Performance [GFLOPS]: " << gflops << std::endl;
    } else {
        std::cout << fft_size << ", " << gflops << ", " << time_n / (inside_repeats * kernel_repeats) << ", "
                  << std::endl;
    }
}

template<unsigned int Arch>
void block_fft_performance() {
    using namespace cufftdx;

    using fft_base = decltype(Block() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() +
                              Precision<float>() + SM<Arch>());

    static constexpr unsigned int elements_per_thread = 8;
    static constexpr unsigned int fft_size            = 512;
    static constexpr unsigned int ffts_per_block      = 1;

    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream))
    benchmark_block_fft<fft_base, fft_size, elements_per_thread, ffts_per_block>(stream, true);
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
}

template<unsigned int Arch>
struct block_fft_performance_functor {
    void operator()() { return block_fft_performance<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<block_fft_performance_functor>();
}
