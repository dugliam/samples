// #include <hip/hip_complex.h>
#include <cufftdx.hpp>
#include <stdio.h>

using namespace cufftdx;

int main() { 
	// base FFT for comparison
	using FFT = decltype(Size<8192>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<700>() + Block());

	printf("Reference memory usage (SM<700>, Size<8192>, ElementsPerThread<32>, Type<fft_type::r2c>)\n");
	printf("\t%d\n", FFT::shared_memory_size);

	// ElementsPerThread = 16 vs ElementsPerThread = 32
	using FFT1_1 = decltype(Size<8192>() + Precision<float>() + Type<fft_type::c2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<16>()
		+ SM<700>() + Block());

	// ElementsPerThread = 8 vs ElementsPerThread = 32
	using FFT1_2 = decltype(Size<8192>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<8>()
		+ SM<700>() + Block());

	printf("ElementsPerThread<x> (SM<700>, Size<8192>, Type<fft_type::r2c>)\n");
	printf("\tElementsPerThread<32>: %d\n", FFT::shared_memory_size);
	printf("\tElementsPerThread<16>: %d\n", FFT1_1::shared_memory_size);
	printf("\tElementsPerThread<8> : %d\n", FFT1_2::shared_memory_size);

	using FFT2_1 = decltype(Size<2048>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<700>() + Block());

	using FFT2_2 = decltype(Size<4096>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<700>() + Block());

	using FFT2_3 = decltype(Size<16384>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<700>() + Block());


	printf("Size<x> (SM<700>, ElementsPerThread<32>, Type<fft_type::r2c>\n");
	printf("\tSize<2048> : %d\n", FFT2_1::shared_memory_size);
	printf("\tSize<4096> : %d\n", FFT2_2::shared_memory_size);
	printf("\tSize<8192> : %d\n", FFT::shared_memory_size);
	printf("\tSize<16384>: %d\n", FFT2_3::shared_memory_size);

	using FFT3_1 = decltype(Size<2048>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<800>() + Block());

	using FFT3_2 = decltype(Size<4096>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<800>() + Block());

	using FFT3_3 = decltype(Size<8192>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<800>() + Block());

	using FFT3_4 = decltype(Size<16384>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<800>() + Block());

	using FFT3_5 = decltype(Size<32768>() + Precision<float>() + Type<fft_type::r2c>() 
		+ Direction<fft_direction::forward>() + FFTsPerBlock<1>() + ElementsPerThread<32>()
		+ SM<800>() + Block());

	printf("Size<x> (SM<800>, ElementsPerThread<32>, Type<fft_type::r2c>\n");
	printf("\tSize<2048> : %d\n", FFT3_1::shared_memory_size);
	printf("\tSize<4096> : %d\n", FFT3_2::shared_memory_size);
	printf("\tSize<8192> : %d\n", FFT3_3::shared_memory_size);
	printf("\tSize<16384>: %d\n", FFT3_4::shared_memory_size);
	printf("\tSize<32768>: %d\n", FFT3_5::shared_memory_size);
	
	return 0;
}

