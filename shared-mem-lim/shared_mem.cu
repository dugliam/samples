
#include <hip/hip_runtime.h>
#include <iostream>


#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, char const* const func, char const* const file,
           int const line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void kernel() { 
    extern __shared__ int temp[];
    
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;

    return;
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const* const file, int const line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() { 
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); 

    // amount of shared memory on the CUDA device
    size_t shared_mem = prop.sharedMemPerMultiprocessor;

    printf("Device name: %s\n", prop.name);
    printf("Amount of shared mem: %lu kB\n", shared_mem);
    printf("Max optin shared mem: %lu kB\n", prop.sharedMemPerBlockOptin);

    // iterate 16 B at a time starting from 1 kB less than the max amount of shared memory
    for(int shared_req = shared_mem - 1024; shared_req < shared_mem; shared_req += 16) {

        printf("Shared mem requested: %lu kB\n", shared_req);

        // increase the amount of shared memory in the kernel to what we request
        CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_req));

        // run the kernel with the requested amount of shared memory
        kernel<<<1, 1, shared_req>>>();

        CHECK_LAST_CUDA_ERROR();
    } 

    printf("Amount of shared mem: %lu kB\n", shared_mem);
    return 0;
}
